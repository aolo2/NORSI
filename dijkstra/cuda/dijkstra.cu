#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <limits>
#include <iomanip>
#include <algorithm>
#include "check.h"

typedef std::pair<unsigned int, unsigned int> graph_dim;

const float inf = std::numeric_limits<float>::infinity();
const unsigned int start = 1; //415;
const unsigned int end = 5	; //330949;

__global__
void bellman_ford(unsigned int iter, unsigned int *starts, unsigned int *ends,
		float *weights, float *dist, unsigned int n) {

	unsigned int from, to;
	float w;

//	printf("n: %d\t index: %d\t stride: %d\n", n, index, stride);

	for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
		from = starts[i];
		to = ends[i];
		w = weights[i];

//		printf("from: %d\t to: %d \t w: %f\n", from, to, w);

		if (dist[from] + w < dist[to]) {
			dist[to] = dist[from] + w;
//			*modif = iter;
		}
	}
}

graph_dim read_file(const std::string &path, unsigned int **starts,
		            unsigned int **ends, float **weights) {

	std::ifstream file(path.c_str());
	std::string line;

	graph_dim size;
	size.first = std::count(std::istreambuf_iterator<char>(file),
				 	 	    std::istreambuf_iterator<char>(), '\n');

	*starts = (unsigned int *) malloc(size.first * sizeof(unsigned int));
	*ends = (unsigned int *) malloc(size.first * sizeof(unsigned int));
	*weights = (float *) malloc(size.first * sizeof(float));

	float weight;
	unsigned int n = 0;
	unsigned int from, to, vertex_num = 0;

	file.close();
	file.open(path.c_str());

	while (file >> from >> to >> weight) {
		vertex_num = std::max(vertex_num, from + 1); // undirected G's are expected
		(*starts)[n] = from;
		(*ends)[n] = to;
		(*weights)[n] = weight;
		++n;
	}

	size.second = vertex_num + 1;

	return size;
}

int main(void) {
	unsigned int *starts = NULL, *ends = NULL, *d_s, *d_e, edge_num, vertex_num;
	float *weights = NULL, *dist, *d_w, *d_dist;

	graph_dim dimensions = read_file("SMALL2_int.sdot", &starts, &ends, &weights);
	edge_num = dimensions.first, vertex_num = dimensions.second;

	dist = (float *) malloc(vertex_num * sizeof(float));

	for (unsigned int i = 0; i < vertex_num; i++) {	dist[i] = inf;}
	dist[start] = 0.0f;

	check(hipMalloc((void **) &d_s, edge_num * sizeof(unsigned int)));
	check(hipMalloc((void **) &d_e, edge_num * sizeof(unsigned int)));
	check(hipMalloc((void **) &d_w, edge_num * sizeof(float)));
	check(hipMalloc((void **) &d_dist, vertex_num * sizeof(float)));

	check(hipMemcpy(d_s, starts, edge_num * sizeof(unsigned int), hipMemcpyHostToDevice));
	check(hipMemcpy(d_e, ends, edge_num * sizeof(unsigned int), hipMemcpyHostToDevice));
	check(hipMemcpy(d_w, weights, edge_num * sizeof(float), hipMemcpyHostToDevice));
	check(hipMemcpy(d_dist, dist, vertex_num * sizeof(float), hipMemcpyHostToDevice));

	int block_size = 512;
	int num_blocks = (edge_num + block_size - 1) / block_size;

	for (unsigned int i = 1; i < edge_num; i++) {
		bellman_ford<<<num_blocks, block_size>>>(i, d_s, d_e, d_w, d_dist, edge_num);
//		check(hipMemcpy(dist, d_dist, vertex_num * sizeof(float), hipMemcpyDeviceToHost));
//		for (unsigned int i = 0; i < vertex_num; i++) { std::cout << dist[i] << " "; }
//		std::cout << std::endl;
	}

	check(hipMemcpy(dist, d_dist, vertex_num * sizeof(float), hipMemcpyDeviceToHost));

	std::cout << dist[end] << std::endl;

	free(starts);
	free(ends);
	free(weights);
	free(dist);

	check(hipFree(d_s));
	check(hipFree(d_e));
	check(hipFree(d_w));
	check(hipFree(d_dist));

	return 0;
}
