#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <limits>
#include <iomanip>
#include <algorithm>
#include "check.h"

__global__
void bellman_ford(unsigned int iter, unsigned int *starts, unsigned int *ends,
		float *weights, float *dist, unsigned int n) {

	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = gridDim.x * blockDim.x;
	unsigned int from, to;
	float w;

//	printf("n: %d\t index: %d\t stride: %d\n", n, index, stride);

	for (unsigned int i = index; i < n; i += stride) {
		w = weights[i];
		from = starts[i];
		to = ends[i];

//		printf("from: %d\t to: %d \t w: %f\n", from, to, w);

		if (dist[from] + w < dist[to]) {
			dist[to] = dist[from] + w;
//			*modif = iter;
		}
	}
}

int main(void) {
	unsigned int modif = 0;
//	*modif = 0;

	std::ifstream r("LEN_int.sdot");
	std::string line;

	const unsigned int edge_num = std::count(std::istreambuf_iterator<char>(r),
			std::istreambuf_iterator<char>(), '\n');

	r.close();
	r.open("LEN_int.sdot");

	const unsigned int start = 0; //415;
	const unsigned int end = 8; //330949;

	unsigned int from, to, vertex_num;
	float weight;

	unsigned int *starts, *ends;
	float *weights, *dist;

	starts = (unsigned int *) malloc(edge_num * sizeof(unsigned int));
	ends = (unsigned int *) malloc(edge_num * sizeof(unsigned int));
	weights = (float *) malloc(edge_num * sizeof(float));

	unsigned int n = 0;
	while (r >> from >> to >> weight) {
		vertex_num = std::max(vertex_num, from + 1); // WRONG FOR DIGRAPGS!!
		starts[n] = from;
		ends[n] = to;
		weights[n] = weight;
		++n;
	}

	dist = (float *) malloc(vertex_num * sizeof(float)); // vertices nums start from 1
	float inf = std::numeric_limits<float>::infinity();

	dist[start] = 0.0f;
	for (unsigned int i = 1; i < vertex_num; i++) {
		dist[i] = inf;
	}

	/*for (unsigned int i = 0; i < edge_num; i++) {
		std::cout << std::left << std::setw(2) << starts[i] << " "
				<< std::setw(2) << ends[i] << " " << weights[i] << std::endl;
	}*/

	unsigned int *d_s, *d_e;
	float *d_w, *d_dist;

	check(hipMalloc((void **) &d_s, edge_num * sizeof(unsigned int)));
	check(hipMalloc((void **) &d_e, edge_num * sizeof(unsigned int)));
	check(hipMalloc((void **) &d_w, edge_num * sizeof(float)));
	check(hipMalloc((void **) &d_dist, vertex_num * sizeof(float)));

	check(hipMemcpy(d_s, starts, edge_num * sizeof(unsigned int), hipMemcpyHostToDevice));
	check(hipMemcpy(d_e, ends, edge_num * sizeof(unsigned int), hipMemcpyHostToDevice));
	check(hipMemcpy(d_w, weights, edge_num * sizeof(float), hipMemcpyHostToDevice));
	check(hipMemcpy(d_dist, dist, vertex_num * sizeof(float), hipMemcpyHostToDevice));

	int block_size = 32;
	int num_blocks = (edge_num + block_size - 1) / block_size;

	for (unsigned int i = 1; i < n; i++) {
		bellman_ford<<<num_blocks, block_size>>>(i, d_s, d_e, d_w, d_dist, edge_num);

//		if (modif == i - 1) {
//			break;
//		}
	}

	check(hipMemcpy(dist, d_dist, vertex_num * sizeof(float), hipMemcpyDeviceToHost));

	for (unsigned int i = 0; i < vertex_num; i++) {
		std::cout << dist[i] << " ";
	}
	std::cout << std::endl;

	check(hipFree(d_s));
	check(hipFree(d_e));
	check(hipFree(d_w));
	check(hipFree(d_dist));

	return 0;
}
